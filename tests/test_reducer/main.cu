#include "hip/hip_runtime.h"
//
// Created by Minghao Guo on 2021/2/25.
//

#include <thrust/device_vector.h>
#include "datatype/image.h"
#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include "helpers/timer.h"
#include <unistd.h>
#include <iomanip>
#include "helpers/tqdm.h"
#include "helpers/random_generator.h"
#include "include/reduce.h"
#include "include/counter.h"
#include "include/arithmetic.h"

int main()
{
    thrust::device_vector<float> vec(4);
    vec[0] = 1.f;
    vec[1] = 2.f;
    vec[2] = 3.f;
    vec[3] = 4.f;

    std::cout << "count 1 = " << count_eq(vec) << std::endl;
    std::cout << "norm2 = " << norm2(vec) << std::endl;
    std::cout << "norm1 = " << norm1(vec) << std::endl;
    std::cout << "norm3 = " << norm(vec, 3.f) << std::endl;
    for (int ind = 0; ind < 4; ind ++)
        std::cout << "pow(2) = " << pow(vec, 2.f)[ind] << std::endl;

    for (int ind = 0; ind < 4; ind ++)
        std::cout << "exp = " << exp(vec)[ind] << std::endl;

    return 0;
}