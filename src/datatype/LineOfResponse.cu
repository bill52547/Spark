#include "hip/hip_runtime.h"
//
// Created by Minghao Guo on 2021/3/4.
//

#include "LineOfResponse.h"
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/gather.h>

using namespace thrust;
using namespace thrust::placeholders;


__host__
device_vector<float> LineOfResponse::GetX1()
{
    device_vector<float> pos_x1(this->num);
    thrust::gather(this->index1.begin(),
                 this->index1.end(),
                 this->lut_pos.begin(),
                 pos_x1.begin());
    return pos_x1;
}

__host__
device_vector<float> LineOfResponse::GetX2()
{
    device_vector<float> pos_x1(this->num);
    thrust::gather(this->index1.begin(),
                   this->index1.end(),
                   this->lut_pos.begin() + MAX_CRYSTAL_ID * 3,
                   pos_x1.begin());
    return pos_x1;
}

__host__
device_vector<float> LineOfResponse::GetY1()
{
    device_vector<float> pos_x1(this->num);
    thrust::gather(this->index1.begin(),
                   this->index1.end(),
                   this->lut_pos.begin() + MAX_CRYSTAL_ID * 1,
                   pos_x1.begin());
    return pos_x1;
}

__host__
device_vector<float> LineOfResponse::GetY2()
{
    device_vector<float> pos_x1(this->num);
    thrust::gather(this->index1.begin(),
                   this->index1.end(),
                   this->lut_pos.begin() + MAX_CRYSTAL_ID * 4,
                   pos_x1.begin());
    return pos_x1;
}

__host__
device_vector<float> LineOfResponse::GetZ1()
{
    device_vector<float> pos_x1(this->num);
    thrust::gather(this->index1.begin(),
                   this->index1.end(),
                   this->lut_pos.begin() + MAX_CRYSTAL_ID * 2,
                   pos_x1.begin());
    return pos_x1;
}

__host__
device_vector<float> LineOfResponse::GetZ2()
{
    device_vector<float> pos_x1(this->num);
    thrust::gather(this->index1.begin(),
                   this->index1.end(),
                   this->lut_pos.begin() + MAX_CRYSTAL_ID * 5,
                   pos_x1.begin());
    return pos_x1;
}

#include "../../src/include/functor.h"
#include "../../src/include/arithmetic.h"

__host__
device_vector<bool> LineOfResponse::IsGapGood()
{
    device_vector<float> xd = absolute<float>(this->GetX2() - this->GetX1());
    device_vector<float> yd = absolute<float>(this->GetY2() - this->GetY1());
    device_vector<float> zd = absolute<float>(this->GetZ2() - this->GetZ1());
    device_vector<float> dd = sqrt<float>(xd * xd + yd * yd + zd * zd);
    device_vector<bool> filt1(this->num);
    thrust::transform(zd.begin(), zd.end(), filt1.begin(), le_functor<float>(LOR_MAX_Z_GAP));
    device_vector<bool> filt2(this->num);
    thrust::transform(dd.begin(), dd.end(), filt2.begin(), ge_functor<float>(LOR_MIN_GAP));
    thrust::transform(filt1.begin(), filt1.end(), filt2.begin(), filt1.begin(), _1 && _2);
    return filt1;
}

#if LOR_WITH_TIME_DIFF
__host__
device_vector<bool> LineOfResponse::InTimeWindow()
{
    device_vector<bool> results(this->num);
    if (this->time_win <= 0 || this->time_diff.size() == 0) {
        thrust::fill(results.begin(), results.end(), true);
        return results;
    }

    device_vector<float> abs_time_diff = absolute<float>(this->time_diff);

    thrust::transform(abs_time_diff.begin(),
                      abs_time_diff.end(),
                      results.begin(),
                      le_functor<float>(this->time_win));
    return results;
}
#else // LOR_WITH_TIME_DIFF
__host__
device_vector<bool> LineOfResponse::InTimeWindow()
{
    device_vector<bool> results(this->num);
    thrust::fill(results.begin(), results.end(), true);
    return results;
}
#endif // LOR_WITH_TIME_DIFF

#if LOR_WITH_ENERGY
__host__
device_vector<bool> LineOfResponse::InEnergyWindow()
{
    device_vector<bool> filt1_low(this->num);
    device_vector<bool> filt1_high(this->num);
    device_vector<bool> filt2_low(this->num);
    device_vector<bool> filt2_high(this->num);
    device_vector<bool> results(this->num);

    if (this->energy1.size() == 0 || this->energy2.size() == 0) {
        thrust::fill(results.begin(), results.end(), true);
        return results;
    }

    thrust::transform(this->energy1.begin(),
                      this->energy1.end(),
                      filt1_low.begin(),
                      ge_functor<float>(this->en_low));

    thrust::transform(this->energy1.begin(),
                      this->energy1.end(),
                      filt1_high.begin(),
                      le_functor<float>(this->en_low));

    thrust::transform(this->energy2.begin(),
                      this->energy2.end(),
                      filt2_low.begin(),
                      ge_functor<float>(this->en_low));

    thrust::transform(this->energy2.begin(),
                      this->energy2.end(),
                      filt2_high.begin(),
                      le_functor<float>(this->en_low));

    thrust::transform(filt1_low.begin(), filt1_low.end(), filt1_high.begin(), filt1_low.begin(), _1 && _2);
    thrust::transform(filt2_low.begin(), filt2_low.end(), filt2_high.begin(), filt2_low.begin(), _1 && _2);
    thrust::transform(filt2_low.begin(), filt2_low.end(), filt2_low.begin(), filt1_low.begin(), _1 && _2);
    return filt1_low;
}
#else // LOR_WITH_ENERGY
__host__
device_vector<bool> LineOfResponse::InEnergyWindow()
{
    device_vector<bool> results(this->num);
    thrust::fill(results.begin(), results.end(), true);
    return results;
}
#endif // LOR_WITH_ENERGY

__host__
device_vector<bool> LineOfResponse::IsAllGood()
{
    device_vector<bool> filt1 = this->IsGapGood();
    device_vector<bool> filt2 = this->InTimeWindow();
    device_vector<bool> filt3 = this->InEnergyWindow();
    thrust::transform(filt1.begin(), filt1.end(), filt2.begin(), filt1.begin(), _1 && _2);
    thrust::transform(filt1.begin(), filt1.end(), filt3.begin(), filt1.begin(), _1 && _2);
    return filt1;
}

struct _is_true
{
    __device__
    bool operator() (const bool x)
    {
        return x;
    }
};

#include <thrust/copy.h>
#include "../include/reduce.h"

__host__
LineOfResponse LineOfResponse::FilterAllGood()
{
    device_vector<bool> stencil = this->IsAllGood();
    int count_all_good = count_true(stencil);
    device_vector<IndexType> new_index1(count_all_good);
    device_vector<IndexType> new_index2(count_all_good);
    device_vector<float> new_time_diff(0);
    device_vector<float> new_table_z(0);
    device_vector<unsigned short> new_energy1(0);
    device_vector<unsigned short> new_energy2(0);

#if LOR_WITH_TIME_DIFF
    if (this->time_diff.size() > 0) {new_time_diff.resize(count_all_good);}
#endif // LOR_WITH_TIME_DIFF

#if LOR_WITH_TABLE_Z
if (this->table_z.size() > 0) {new_table_z.resize(count_all_good);}
#endif // LOR_WITH_TABLE_Z

#if LOR_WITH_ENERGY
    if (this->energy1.size() > 0 && this->energy2.size() > 0)
    {
        new_energy1.resize(count_all_good);
        new_energy2.resize(count_all_good);
    }
#endif // LOR_WITH_ENERGY
#if LOR_WITH_TIME_DIFF
    LineOfResponse new_lor = {new_index1, new_index2, new_time_diff};
#else // LOR_WITH_TIME_DIFF
    LineOfResponse new_lor = {new_index1, new_index2};
#endif // LOR_WITH_TIME_DIFF
    new_lor.SetLutPos(this->lut_pos);
#if LOR_WITH_TABLE_Z
    new_lor.SetTableZ(new_table_z);
#endif // LOR_WITH_TABLE_Z
#if LOR_WITH_ENERGY
    new_lor.SetEnergy(new_energy1, new_energy2);
#endif // LOR_WITH_ENERGY
    return new_lor;
}

#include <nlohmann/json.hpp>
#include <iomanip>
#include <fstream>
#include "../../src/include/rawdata_io.h"

using json = nlohmann::json;

void LineOfResponse::write_json(const std::string & path) const {
    size_t lastindex = path.find_last_of(".");
    std::string raw_path = path.substr(0, lastindex);
    json j;
    j["LOR_WITH_INDEX"] = LOR_WITH_INDEX;
    j["LOR_WITH_TIME_DIFF"] = LOR_WITH_TIME_DIFF;
    j["LOR_WITH_TABLE_Z"] = LOR_WITH_TABLE_Z;
    j["LOR_WITH_ENERGY"] = LOR_WITH_ENERGY;
    j["num"] = this->num;

    std::string data_path = raw_path + "_data.bin";
    std::ofstream FILE(data_path, std::ios::out | std::ofstream::binary);
    j["data_path"] = data_path;


#if LOR_WITH_INDEX
    j["LOR_MIN_GAP"] = LOR_MIN_GAP;
    j["LOR_MAX_Z_GAP"] = LOR_MAX_Z_GAP;
    std::string lut_pos_path = raw_path + "_lut_pos.bin";
    // std::string index1_path = raw_path + "_index1.bin";
    // std::string index2_path = raw_path + "_index2.bin";
    // std::ofstream FILE1(index1_path, std::ios::out | std::ofstream::binary);
    vector_writer(FILE, this->index1);
    // j["index1_path"] = index1_path;

    // std::ofstream FILE2(index2_path, std::ios::out | std::ofstream::binary);
    vector_writer(FILE, this->index2);
    // j["index2_path"] = index2_path;

    std::ofstream FILE3(lut_pos_path, std::ios::out | std::ofstream::binary);
    vector_writer(FILE3, this->lut_pos);
    j["lut_pos_path"] = lut_pos_path;
#endif // LOR_WITH_INDEX

#if LOR_WITH_TIME_DIFF
    j["time_win"] = time_win;
    // std::string time_diff_path = raw_path + "_time_diff.bin";
    // std::ofstream FILE(time_diff_path, std::ios::out | std::ofstream::binary);
    vector_writer(FILE, this->time_diff);
    // j["time_diff_path"] = time_diff_path;
#endif // LOR_WITH_TIME_DIFF

#if LOR_WITH_TABLE_Z
    // std::string table_z_path = raw_path + "_table_z.bin";
    // std::ofstream FILE(table_z_path, std::ios::out | std::ofstream::binary);
    vector_writer(FILE, this->table_z);
    // j["table_z_path"] = table_z_path;
#endif // LOR_WITH_TABLE_Z

#if LOR_WITH_ENERGY
    j["en_low"] = ENERGY_LOW;
    j["en_high"] = ENERGY_HIGH;

    // std::string energy1_path = raw_path + "_energy1.bin";
    // std::ofstream FILE1(energy1_path, std::ios::out | std::ofstream::binary);
    vector_writer(FILE, this->energy1);
    // j["energy1_path"] = energy1_path;

    // std::string energy2_path = raw_path + "_energy2.bin";
    // std::ofstream FILE2(energy2_path, std::ios::out | std::ofstream::binary);
    vector_writer(FILE, this->energy2);
    // j["energy2_path"] = energy2_path;
#endif // LOR_WITH_ENERGY

    std::ofstream o(path);
    o << std::setw(4) << j << std::endl;
}

LineOfResponse LineOfResponse::read_json(const std::string & filename)
{
    std::ifstream i(filename);
    json j;
    i >> j;
    assert(j["LOR_WITH_INDEX"] == LOR_WITH_INDEX);

    int num = j["num"];
    LineOfResponse lor(num);
    std::string data_path = j["data_path"];
    std::ifstream FILE(data_path, std::ios::in | std::ifstream::binary);

#if LOR_WITH_INDEX
    // std::string index1_path = j["index1_path"];
    // std::ifstream FILE1(index1_path, std::ios::in | std::ifstream::binary);
    device_vector<IndexType> index1 = vector_reader<IndexType>(FILE, num);

    // std::string index2_path = j["index2_path"];
    // std::ifstream FILE2(index2_path, std::ios::in | std::ifstream::binary);
    device_vector<IndexType> index2 = vector_reader<IndexType>(FILE, num);

    std::string lut_pos_path = j["lut_pos_path"];
    std::ifstream FILE3(lut_pos_path, std::ios::in | std::ifstream::binary);
    device_vector<float> lut_pos = vector_reader<float>(FILE3, num * 6);
#endif // LOR_WITH_INDEX

    assert(j["LOR_WITH_TIME_DIFF"] == LOR_WITH_TIME_DIFF);
#if LOR_WITH_TIME_DIFF
    // std::string time_diff_path = j["time_diff_path"];
    // std::ifstream FILE(time_diff_path, std::ios::in | std::ifstream::binary);
    device_vector<float> time_diff = vector_reader<float>(FILE, num);
    lor = LineOfResponse(index1, index2, time_diff);
#else
    lor = LineOfResponse(index1, index2);
#endif // LOR_WITH_TIME_DIFF
    lor.SetLutPos(lut_pos);

    assert(j["LOR_WITH_TABLE_Z"] == LOR_WITH_TABLE_Z);

#if LOR_WITH_TABLE_Z
    // std::string table_z_path = j["table_z_path"];
    // std::ifstream FILE(table_z_path, std::ios::in | std::ifstream::binary);
    device_vector<float> table_z = vector_reader<float>(FILE, num);
    lor.SetTableZ(table_z);
#endif // LOR_WITH_TABLE_Z

    assert(j["LOR_WITH_ENERGY"] == LOR_WITH_ENERGY);
#if LOR_WITH_ENERGY
    // std::string energy1_path = j["energy1_path"];
    // std::ifstream FILE1(energy1_path, std::ios::in | std::ifstream::binary);
    device_vector<unsigned short> energy1 = vector_reader<unsigned short>(FILE, num);

    // std::string energy2_path = j["energy2_path"];
    // std::ifstream FILE2(energy2_path, std::ios::in | std::ifstream::binary);
    device_vector<unsigned short> energy2 = vector_reader<unsigned short>(FILE, num);
    lor.SetEnergy(energy1, energy2);
#endif // LOR_WITH_ENERGY

    return lor;
}
