#include "hip/hip_runtime.h"
//
// Created by Minghao Guo on 2021/3/5.
//

#include "BackProjector.h"
#include "../config/config.h"
#include <iostream>

#define ABS(x) ((x > 0) ? x : -(x))
#define MAX(a, b) (((a) > (b)) ? a : b)
#define MIN(a, b) (((a) < (b)) ? a : b)
const float PI = 3.1415926;
const float eps_ = 0.00443;
#define IndexType LOR_INDEX_TYPE
#define numCry MAX_CRYSTAL_ID
#include "../helpers/cuda_safe_call.h"
struct DImage
{
    int nx, ny, nz;
    float *affine;
    float *data;
};

struct DListmode
{
    int num;
    float *data;
    IndexType *index1;
    IndexType *index2;
    float *time_diff;
    float *lut_cry;
};

__device__
float cal_dist(const float x1, const float y1, const float z1,
                          const float x2, const float y2, const float z2) {
    return (float)sqrtf((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) +
                        (z1 - z2) * (z1 - z2));
}

__device__ void
backproject_device(float *image, const float vproj, const float x1l,
                   const float x1r, const float y1l, const float y1r,
                   const float z1b, const float z1f, const float x2l,
                   const float x2r, const float y2l, const float y2r,
                   const float z2b, const float z2f,
                   const int nx, const int ny, const int nz,
                   const float dx, const float dy, const float dz,
                   const float tof_value, const float tof_resolution) {
    const float x1c = (x1l + x1r) / 2;
    const float y1c = (y1l + y1r) / 2;
    const float z1c = (z1f + z1b) / 2;
    const float x2c = (x2l + x2r) / 2;
    const float y2c = (y2l + y2r) / 2;
    const float z2c = (z2f + z2b) / 2;

    const float tof_sigma = tof_resolution / 2.355 * 0.3 / 2;
    const float over_tof_sigma2 = 0.5 / tof_sigma / tof_sigma;
    const float over_sqrt_2pi_tof_sigma2 =
            1 / sqrt(2.0 * PI * tof_sigma * tof_sigma);
    const float L = cal_dist(x1c, y1c, z1c, x2c, y2c, z2c);
    const float ratio1 = (1.0 - tof_value * 0.3 / L) / 2;
    float tof_weight;
//
    if (ABS(x1c - x2c) > ABS(y1c - y2c)) {
        float tan_alpha = (y1c - y2c) / (x1c - x2c);
        float tan_gamma = (z1c - z2c) / (x1c - x2c);
        float inv_cos_alpha = (float)sqrtf(1 + tan_alpha * tan_alpha);
        float inv_cos_gamma = (float)sqrtf(1 + tan_gamma * tan_gamma);
        for (int ix = 0; ix < nx; ix++) {
            float xc, ratio, cylr, cyrl, czb, czf, yi1, yi2, zi1, zi2;
            int Yi1, Yi2, Zi1, Zi2;
            xc = ((float)ix + 0.5f - (float)nx / 2) * dx;
            float weight = inv_cos_gamma * inv_cos_alpha;
            // float weight = 1;
            if (tof_sigma > 0) {
                float d2_tof = ((xc - x1c) / (x2c - x1c) - ratio1) * L;
                float t2 = -d2_tof * d2_tof * over_tof_sigma2;
                tof_weight =
                        __expf(t2) * over_sqrt_2pi_tof_sigma2 * dx * L / ABS(x1c - x2c);
                if (tof_weight < eps_) {
                    continue;
                }
                weight *= tof_weight;
            }
            ratio = (y2r - y1l) / (x2r - x1l);
            cylr = ratio * (xc - x1l) + y1l + ny / 2 * dy;

            ratio = (y2l - y1r) / (x2l - x1r);
            cyrl = ratio * (xc - x1r) + y1r + ny / 2 * dy;

            yi1 = MIN(cyrl, cylr);
            Yi1 = (int)floorf(yi1 / dy);
            yi2 = MAX(cyrl, cylr);
            Yi2 = (int)floorf(yi2 / dy);

            ratio = (z2b - z1b) / (x2c - x1c);
            czb = ratio * (xc - x1c) + z1b + nz / 2 * dz;
            ratio = (z2f - z1f) / (x2c - x1c);
            czf = ratio * (xc - x1c) + z1f + nz / 2 * dz;

            zi1 = MIN(czb, czf);
            Zi1 = (int)floorf(zi1 / dz);
            zi2 = MAX(czb, czf);
            Zi2 = (int)floorf(zi2 / dz);
            float wy, wz;
            for (int iy = MAX(0, Yi1); iy <= MIN(ny - 1, Yi2); iy++)
            {
                wy = MIN((iy + 1.0f) * dy, yi2) - MAX((iy + 0.0f) * dy, yi1);
                wy /= (yi2 - yi1);
                for (int iz = MAX(0, Zi1); iz <= MIN(nz - 1, Zi2); iz++) {
                    wz = MIN((iz + 1.0f) * dz, zi2) - MAX((iz + 0.0f) * dz, zi1);
                    wz /= (zi2 - zi1);
                    if (iy >= 0 && iy < ny && iz >= 0 && iz < nz)
                        atomicAdd(image + ix + iy * nx + iz * nx * ny,
                                  vproj * wy * wz * weight);
                }
            }
        }
    } else {
        float tan_alpha = (x1c - x2c) / (y1c - y2c);
        float tan_gamma = (z1c - z2c) / (y1c - y2c);
        float inv_cos_alpha = (float)sqrtf(1 + tan_alpha * tan_alpha);
        float inv_cos_gamma = (float)sqrtf(1 + tan_gamma * tan_gamma);
        for (int iy = 0; iy < ny; iy++) {
            float yc, ratio, cxlr, cxrl, czb, czf, xi1, xi2, zi1, zi2;
            int Xi1, Xi2, Zi1, Zi2;
            yc = ((float)iy + 0.5f - (float)ny / 2) * dy;
            float weight = inv_cos_gamma * inv_cos_alpha;
            if (tof_sigma > 0) {
                float d2_tof = ((yc - y1c) / (y2c - y1c) - ratio1) * L;
                float t2 = -d2_tof * d2_tof * over_tof_sigma2;
                tof_weight =
                        __expf(t2) * over_sqrt_2pi_tof_sigma2 * dy * L / ABS(y1c - y2c);
                if (tof_weight < eps_) {
                    continue;
                }
                weight *= tof_weight;
            }
            ratio = (x2r - x1l) / (y2r - y1l);
            cxlr = ratio * (yc - y1l) + x1l + nx / 2 * dx;

            ratio = (x2l - x1r) / (y2l - y1r);
            cxrl = ratio * (yc - y1r) + x1r + nx / 2 * dx;

            xi1 = MIN(cxrl, cxlr);
            Xi1 = (int)floorf(xi1 / dx);
            xi2 = MAX(cxrl, cxlr);
            Xi2 = (int)floorf(xi2 / dx);

            ratio = (z2b - z1b) / (y2c - y1c);
            czb = ratio * (yc - y1c) + z1b + nz / 2 * dz;
            ratio = (z2f - z1f) / (y2c - y1c);
            czf = ratio * (yc - y1c) + z1f + nz / 2 * dz;

            zi1 = MIN(czb, czf);
            Zi1 = (int)floorf(zi1 / dz);
            zi2 = MAX(czb, czf);
            Zi2 = (int)floorf(zi2 / dz);
            float wx, wz;
            for (int ix = MAX(0, Xi1); ix <= MIN(nx - 1, Xi2); ix++)
            {
                wx = MIN((ix + 1.0f) * dx, xi2) - MAX((ix + 0.0f) * dx, xi1);
                wx /= (xi2 - xi1);
                for (int iz = MAX(0, Zi1); iz <= MIN(nz - 1, Zi2); iz++) {
                    wz = MIN((iz + 1.0f) * dz, zi2) - MAX((iz + 0.0f) * dz, zi1);
                    wz /= (zi2 - zi1);
                        atomicAdd(image + ix + iy * nx + iz * nx * ny,
                              vproj * wx * wz * weight);
                }
            }
        }
    }
}

__global__ void bproject(DImage image,
        const DListmode listmode,
        const float ctr = -1.f)
{
    int step = blockDim.x * gridDim.x;

    for (int tid = blockIdx.x * blockDim.x + threadIdx.x;
    tid < (listmode.num + step); tid += step) {
        if (tid >= listmode.num) {
            return;
        }
        const int nx = image.nx, ny = image.ny, nz = image.nz;
        const float dx = image.affine[0] / nx, dy = image.affine[5] / ny, dz = image.affine[10] / nz;
        const float cx = image.affine[4], cy = image.affine[8], cz = image.affine[12];
        int id1 = listmode.index1[tid], id2 = listmode.index2[tid];
        float time_diff_ = 0.f;
        if (ctr > 0) {time_diff_ = listmode.time_diff[tid];}
        if (tid >= nx * ny * nz) {return;}
        backproject_device(image.data,
                listmode.data[tid],
                listmode.lut_cry[id1 + 0 * numCry] + cx,
                listmode.lut_cry[id1 + 3 * numCry] + cx,
                listmode.lut_cry[id1 + 1 * numCry] + cy,
                listmode.lut_cry[id1 + 4 * numCry] + cy,
                listmode.lut_cry[id1 + 2 * numCry] + cz,
                listmode.lut_cry[id1 + 5 * numCry] + cz,
                listmode.lut_cry[id2 + 0 * numCry] + cx,
                listmode.lut_cry[id2 + 3 * numCry] + cx,
                listmode.lut_cry[id2 + 1 * numCry] + cy,
                listmode.lut_cry[id2 + 4 * numCry] + cy,
                listmode.lut_cry[id2 + 2 * numCry] + cz,
                listmode.lut_cry[id2 + 5 * numCry] + cz,
                nx, ny, nz, dx, dy, dz,
                time_diff_,
                ctr);
    }
}


Image BackProjector::operator() (Listmode & listmode)
{
    device_vector<float> out_data(this->image_template.GetDataNumber());
    DImage d_image = {this->image_template.GetShape().x,
                       this->image_template.GetShape().y,
                       this->image_template.GetShape().z,
                       thrust::raw_pointer_cast(this->image_template.GetAffine().data.data()),
                       thrust::raw_pointer_cast(out_data.data())};

    DListmode d_listmode = {
        listmode.GetNum(),
        listmode.GetDataRawPtr(),
        listmode.GetIndex1RawPtr(),
        listmode.GetIndex2RawPtr(),
        listmode.GetTimeDiffRawPtr(),
        listmode.GetLutPosRawPtr()
    };

    float *out_data_ptr = raw_pointer_cast(out_data.data());
    bproject<<<listmode.GetNum() / 512 + 1 ,512>>>(d_image, d_listmode, this->CTR);
    CHECK_LAUNCH_ERROR();
    return this->image_template.UpdateData(out_data);
}
